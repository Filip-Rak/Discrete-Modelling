#include "AutomatonCUDA.h"

// Constructor: Set up device memory
AutomatonCUDA::AutomatonCUDA(int width, int height) : 
	width(width), height(height)
{
	// Check CUDA availability
	cuda_available = check_CUDA_availability();;

	// Set memory as NULL if CUDA not present
	if (!cuda_available)
	{
		// d_cells = nullptr;
		return;
	}

	allocate_memory();
}

// Destructor: Free device memory
AutomatonCUDA::~AutomatonCUDA()
{
	if (!cuda_available)
		return;
	
	// Free all allocated memory
	if (d_initial_grid)
		hipFreeArray(d_initial_grid);

	if (d_outputs_tex)
		hipFreeArray(d_outputs_tex);

	if (d_outputs)
		hipFree(d_outputs);

	if (d_inputs)
		hipFree(d_inputs);

	if (results)
		hipHostFree(results);

}

// Copy initial state to GPU
void AutomatonCUDA::send(const uint16_t* h_cells)
{
	if (!cuda_available)
		return;

	// Find the size of the data to copy
	size_t size = width * height * sizeof(uint16_t);

	// Copy the host data to the CUDA array bound to the initial grid
	hipError_t return_code = hipMemcpyToArray(d_initial_grid, 0, 0, h_cells, size, hipMemcpyHostToDevice);
}

// Perform GPU based update
void AutomatonCUDA::update()
{
	if (!cuda_available)
		return;

	// Call kernels
}

// Copy GPU's state to CPU
void AutomatonCUDA::retrieve(uint16_t* h_cells)
{
	if (!cuda_available)
		return;
}

/* Private Methods */

// Check if CUDA is available on this device
bool AutomatonCUDA::check_CUDA_availability()
{
	int device_count = 0;
	hipError_t err = hipGetDeviceCount(&device_count);

	if (err != hipSuccess || device_count == 0)
	{
		return false;
	}

	std::cout << "CUDA Available with: " << device_count << " device(s).\n";
	return true;
}

void AutomatonCUDA::allocate_memory()
{
	// Set texture memory as 16 bit
	channel_desc = hipCreateChannelDesc<uint16_t>();

	// Allocate CUDA array for initial grid (read-only texture memory)
	size_t alloc_size = width * height * sizeof(uint16_t);
	hipError_t return_code = hipMallocArray(&d_initial_grid, &channel_desc, width, height);
	print_malloc_failure(return_code, "initial grid (read-only)", alloc_size);

	// Allocate device memory for outputs
	return_code = hipMalloc(&d_outputs, alloc_size);
	print_malloc_failure(return_code, "outputs", alloc_size);

	// Allocate CUDA array for outputs as read-only texture
	return_code = hipMallocArray(&d_outputs_tex, &channel_desc, width, height);
	print_malloc_failure(return_code, "outputs (read-only)", alloc_size);

	// Allocate device memory for inputs
	alloc_size = width * height * 5 * sizeof(uint16_t);
	return_code = hipMalloc(&d_inputs, alloc_size);
	print_malloc_failure(return_code, "inputs", alloc_size);
	
	// Allocate pinned host memory for results
	return_code = hipHostAlloc(&results, alloc_size, hipHostMallocMapped);
	print_malloc_failure(return_code, "results", alloc_size);
}

void AutomatonCUDA::print_malloc_failure(hipError_t success_code, std::string name, int size)
{
	if (success_code != hipSuccess)
		std::cerr << "CUDA malloc for " + name + " failed for size " << size << ": " << hipGetErrorString(success_code) << "\n";
}
